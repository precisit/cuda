#include "hip/hip_runtime.h"
#include "cuda_heat_eq.h"

//extern float *GLOBAL_A(nullptr);
extern int GLOBAL_N = 0;
extern float* U_0(NULL);


void prvec(float *A, int n){
	for(int x=0; x<n; x++){
		std::cout<<A[x]<<"   ";
	}
}

void prmat(float *A, int n){
	for(int x=0; x<n; x++){
		for(int y=0; y<n; y++){
			std::cout<<A[x+y*n]<<"   ";
		}
		std::cout<<std::endl;
	}
}

//Solves Ax=b for x. (A is n-by-n and also symmetric).
//Based on CG.
void matSolve(const float *A, float *x, const float *b, const int n, const float absTol, hipblasHandle_t handle){
	//Stolen from here: https://en.wikipedia.org/wiki/Conjugate_gradient_method
	float *r, *p, *Ap;
	float r_norm_old, r_norm;
	
	hipMalloc ((void**)&r, n*sizeof(*r));
	hipMalloc ((void**)&p, n*sizeof(*p));
	hipMalloc ((void**)&Ap, n*sizeof(*Ap));
	
	
	//Kolla om man får nån speed-up av att flytta dessa till GPUn. FIX!
	float alpha, beta, negative_alpha;
	
	//Kolla om man får nån speed-up av att flytta dessa till GPUn. FIX!
	float one = 1.0f;
	float negative_one = -1.0f;
	float zero = 0.0f;
	
	//r = b-A*x
	hipblasScopy(handle,n, b, 1, r,1);
	hipblasSgemv(handle,HIPBLAS_OP_N,n,n,&negative_one, A,n,x,1,&one,r, 1);
	
	//define r_norm_old
	hipblasSdot(handle,n,r,1,r,1,&r_norm_old);
	//hipblasSnrm2(handle,n,r,1, &r_norm_old);
	
	//p = r
	hipblasScopy(handle, n, r, 1, p,1);
	
	for(int i=0; i<n; i++){
		std::cout<<"hej?"<<std::endl;
	
		//Ap = A*p
		hipblasSgemv(handle,HIPBLAS_OP_N,n,n,&one, A,n,p,1,&zero,Ap, 1);
		
		//update alpha
		hipblasSdot(handle,n,p,1,Ap,1,&alpha);
		alpha = r_norm_old/alpha;
		assert(alpha>=0.0f);
		
		//x = x + alpha*p
		hipblasSaxpy(handle,n,&alpha, p,1, x, 1);
		
		//r = r - alpha*A*p (=r-alpha*Ap. FIX! hipblasSaxpy() borde funka)
		negative_alpha = - alpha;
		hipblasSgemv(handle,HIPBLAS_OP_N,n,n,&negative_alpha, A,n,p,1,&one,r, 1);
	
		//if r small: break
		hipblasSdot(handle,n,r,1,r,1,&r_norm);
		if(r_norm < absTol){
			break;
		}
		else{
			assert(r_norm != 0.f);
		}
		
		//update beta
		beta = r_norm/r_norm_old;
		
		//p = r + beta*p
		 hipblasSscal(handle, n,&beta, p,1);
		 hipblasSaxpy(handle, n, &one, r, 1, p, 1);
		 
		 r_norm_old = r_norm;
	}
		
	hipFree(r);
	hipFree(p);
	hipFree(Ap);
}

int dist(const int x1, const int y1, const int x2, const int y2){
	//Returns the distance (measured in the chess king's norm) between 2 points.
	return abs(x1-x2)+abs(y1-y2);
}

void setStiffMat(float *A, const class Element *phi, const int n){
	const float h = phi->h; 
	int d;

	for(int x1=0; x1<n; x1++){
		for(int y1=0; y1<n; y1++){
			for(int x2=0; x2<n; x2++){
				for(int y2=0; y2<n; y2++){
					d = dist(x1,y1,x2,y2);
					if(d == 0){
						A[(x1*n+y1)*n*n+(x2*n+y2)] = 4.0f/(h*h); //fett ad hoc! FIX! //O dessutom stämmer det nog inte. Eller?
					}
				}
			}
		}
	}
}

void setMassMat(float *M, const class Element *phi, const int n){
	//This is stupid and stuff. And shouldn't be done on the CPU. Oh, well.
	//But that'll be fixed later. 
	//(And also it's done ad hoc for hat funcs)
	
	
	//Antar att alla har samma storlek. FIX!
	const float h = phi->h;
	int d;
	
	for(int x1=0; x1<n; x1++){
		for(int y1=0; y1<n; y1++){
			for(int x2=0; x2<n; x2++){
				for(int y2=0; y2<n; y2++){
					d = dist(x1,y1,x2,y2);
					if(d == 0){
						M[(x1*n+y1)*n*n+(x2*n+y2)] = h*h/3.0f; //wolfram alpha said so. http://www.wolframalpha.com/input/?i=4*int_0%5Eh+int_0%5E%28h-x%29+%281-%28x%2By%29%2Fh%29%5E2+dy+dx
					}
					else if(d == 1){
						M[(x1*n+y1)*n*n+(x2*n+y2)] = h*h/24.0f; //http://www.wolframalpha.com/input/?i=4*int_0%5E%28h%2F2%29+int_0%5E%28x%29+%281-%28x%2By%29%2Fh%29*%281-%28-%28x-h%29%2By%29%2Fh%29+dy+dx
					}
				}
			}
		}
	}
}

void waveGal(float *U_0, const int n, const float dt, const float endTime, const float tol){


	::U_0 = U_0;


	::GLOBAL_N = n;
	assert(endTime>0.0f);
	assert(n>0);
	
	float t=0;
	
	//Setup the Element (should be a wavelet, but isn't yet. FIX!)
	Element phi = Element(0.0f, 0.0f, 1.0f/((float)(n-1)));
	float h = phi.h;
	
	/*
	//For debugging. Remove. FIX!
	float *Q;
	Q = (float*) calloc (n*n*n*n,sizeof(*Q));
	*/
	
	//Setup mass and stiffness matrix;
	float *M,*A;
	M = (float*) calloc (n*n*n*n,sizeof(*M));
	A = (float*) calloc (n*n*n*n,sizeof(*A));
	setMassMat(M, &phi,n);
	setStiffMat(A, &phi,n);
	
	//Move everything to the GPU
	float *devM, *devA, *devU, *devB;
	
    hipblasHandle_t handle;
    hipblasCreate(&handle); 
	
	hipMalloc ((void**)&devM, n*n*n*n*sizeof(*devM));
	hipMalloc ((void**)&devA, n*n*n*n*sizeof(*devA));
	hipMalloc ((void**)&devU, n*n*sizeof(*devU));
	hipMalloc ((void**)&devB, n*n*sizeof(*devB));
	
	hipblasSetMatrix(n*n,n*n, sizeof(*A), A, n*n, devA, n*n);
	hipblasSetMatrix(n*n,n*n, sizeof(*M), M, n*n, devM, n*n);
	hipblasSetVector(n*n, sizeof(*U_0), U_0, 1, devU, 1);
	
	free(A); free(M);
	
	//Integrate forward through time
	//Antar att M och A är konstanta. Vilket de är än så länge. 
	//Adaptive grids kommer dessvärre ändra det. 
	//FIX!
	
	//Reassign A as A := M+k*A; //FIX!
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &h, devA, n, 0, devM, n, devA, n);

	const float one = 1.0f;
	float zero = 0.0f;
	
	display();
	while(t<endTime){
		//b = M_old*U_old;
		hipblasSgemv(handle, HIPBLAS_OP_N, n*n, n*n,&one, devM, n*n, devU, 1, &zero, devB, 1); //M är förmodligen bandmarix. Använd annan funk!! //FIX
		
		//Solve A*u = b
		matSolve(devA, devU, devB, n*n, 0.1f, handle);
		
		hipblasGetVector(n*n, sizeof(float), devU, 1, U_0,1); //Det här är segt. FIX!

		::U_0 = U_0;
		
		//måla här.
		display();
		
		t += dt;
		return;
	}
	
	hipblasGetVector(n*n, sizeof(*U_0), devU, 1, U_0, 1);
	
	hipFree(devM);
	hipFree(devA);
	hipFree(devU);
	hipFree(devB);
	
	hipblasDestroy(handle);
	
}






#ifdef __APPLE_CC__
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif
#include <time.h>


void color(int x, int array[]){
		if (x==0){
			//std::cout<<"red"<<std::endl;
			array[0] = 1.0; array[1] = 0.0; array[2] = 0.0;	
						
		}			
		else if (x==1){
			std::cout<<"not red"<<std::endl;
			array[0] = 1.0; array[1] = 0.5; array[2] = 0.0;
			
		}
		else if (x==2){
			std::cout<<"not red"<<std::endl;
			array[0] = 1.0; array[1] = 1.0; array[2] = 0.0;
			
		}
		else if (x==3){
			std::cout<<"not red"<<std::endl;
			array[0] = 0.5; array[1] = 1.0; array[2] = 0.0;
			
		}
		else if (x==4){
			std::cout<<"not red"<<std::endl;
			array[0] = 0.5; array[1] = 1.0; array[2] = 0.5;
			
		}
		else if (x==5){
			std::cout<<"not red"<<std::endl;
			array[0] = 0.0; array[1] = 0.5; array[2] = 0.0;
			
		}
		else if (x==6){
			std::cout<<"not red"<<std::endl;
			array[0] = 0.0; array[1] = 1.0; array[2] = 1.0;
			
		}
		else {
			std::cout<<"not red"<<std::endl;
			array[0] = 0.0; array[1] = 0.0; array[2] = 1.0;
		}
}

void display() {
	glClear(GL_COLOR_BUFFER_BIT);			//clear buffers to preset values (Indicates the buffers currently enabled for color writing.)
	//int number = 100;
	int r;
	int g;
	int b;	
	int colorarray[3];
	//float m [number];
	
	int n = ::GLOBAL_N;
	
	int val;
	float val_f;
	
	glBegin(GL_QUADS);
	for (int x = 0; x < n; x ++){
		for (int y = 0; y < n; y ++){
		//std::cout<<"tjeeeeeeeeeena!"<<std::endl;
			glVertex2f(x, y);  
			glVertex2f(x+1, y);
			glVertex2f(x+1, y+1);
			glVertex2f(x, y+1);

			//for (int k = 0; k < number; k++) {
				//int random = rand()%8;
				
				val_f = ::U_0[x*n+y];
				val = (int) (val_f*7.0f);
				//m[j] = random; 				
				color(val, colorarray);
				r = colorarray[0];
				g = colorarray[1];
				b = colorarray[2];
				glColor3f(r, g, b);
				//}
			//glMultMatrixf(m);
		}
	
			
	}
	
	//std::cout<<"klaaaaar"<<std::endl;
	glEnd();
	glFlush();
			
}
		
	

	
	
	

/*glBegin(GL_QUADS);
	for (GLfloat i = 0; i<= 100; i ++){
		for (GLfloat j = 0; j<= 100; j ++){
		glVertex2f(i*2, j*2);
		glVertex2f(i, j);
		color(4, colorarray);
			r = colorarray[0];
			g = colorarray[1];
			b = colorarray[2];
			glColor3f(r, g, b);
	}
	}
	glEnd();*/




void init() {
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, ::GLOBAL_N, ::GLOBAL_N, 0, -1, 1);
}

void timer(int v) {
	glutDisplayFunc(display);
	glutPostRedisplay();
	glutTimerFunc(v, timer, v);
}


/*
void reshape() {
}*/

/*

int main(int argc, char** argv) {

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB); 	
	glutInitWindowPosition(200, 200);		
	glutInitWindowSize(800, 800);		
	glutCreateWindow("Test of Grid");
	init();			
	//glutDisplayFunc(display);
	//glutReshapeFunc(reshape);
	glutTimerFunc(100, timer, 0);			
	glutMainLoop();				
}

*/













/*
int main(){
	int n = 3;
	float* U_0;
	U_0 = (float*) calloc (n*n,sizeof(*U_0));
	
	U_0[n+1] = 1.0f;

	waveGal(U_0, n, 0.02f, 0.04f, 0.1f);

	for(int i=0; i<n; i++){
		std::cout<<U_0[i]<<std::endl;
	}

	std::cout<<"it works?"<<std::endl;
	
	return 0;
}

*/
