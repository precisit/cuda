#include "hip/hip_runtime.h"
//#include <iostream>
//#include "node.cpp"
//#include "parameters.h"
//#include "wavelet.cu"
//#include <cmath>
#include "wavelet_d_cuda.h"



__device__ float checkInterPol(float p5, float dot){

 	float p;

 	p = p5 - dot;
 	
 	/*if (p<0){

 		p = p*(-1);
 	}*/

 	return p;	

}


__device__ float interpolDotEdge(float p1, float p2){
//float interpolDotEdge(float p1, float p2){

 	float p;

 	p = (p1 + p2)/2.0;

 	return p;
 
 }

__device__ float interpolDotMiddle(float p1, float p2, float p3, float p4){
//float interpolDotMiddle(float p1, float p2, float p3, float p4){

 	float p;

 	p = (p1 + p2 + p3 + p4)/4.0;

 	return p;
 	
 }

__device__ int pow(int layer, int steplen){

 	int s = 1;

 	for(int i = 0; i<layer; i++){
 		
 		s = s*steplen;
 	}	

 	return s;
 }



__global__ void wavelet_d_kernal(Node* matrix, Node* array, int row, int colum, int countTrue, int layers){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;

	for(int i = 0; i<countTrue; i++){

		if(matrix[idx*colum + idy].x_index_global == array[i].x_index_global && matrix[idx*colum + idy].y_index_global == array[i].y_index_global){

			matrix[idx*colum + idy] = array[i];
		}
	}

	for(int i = layers; i>0; i--){

		int step = pow(i, 2);

		if(matrix[idx*colum + idy].layer = i){
			
			
						
			if(matrix[idx*colum + idy].isPicked == true && (matrix[idx*colum + idy].x_index_global <= (row-step-1)) && (matrix[idx*colum + idy].y_index_global <= (colum-step-1))){

				float p1 = matrix[idx*colum + idy].vort;
				float p2 = matrix[(idx+step)*colum + idy].vort;
				float p3 = matrix[(idx+step)*colum + idy+step].vort;
				float p4 = matrix[idx*colum + idy+step].vort;
				//float p5 = matrix[(idx + step/2)*colum + idy + step/2].vort;
									
				int x1 = matrix[idx*colum + idy].x_index_global;
				int x2 = matrix[(idx+step)*colum + idy].x_index_global;				
				int x3 = matrix[(idx+step)*colum + idy+step].x_index_global;
				int x4 = matrix[idx*colum + idy+step].x_index_global;	
				
				int y1 = matrix[idx*colum + idy].y_index_global;			
				int y2 = matrix[(idx+step)*colum + idy].y_index_global;
				int y3 = matrix[(idx+step)*colum + idy+step].y_index_global;
				int y4 = matrix[idx*colum + idy+step].y_index_global;			
				
				if (matrix[((x1 + x2)/2)*colum + (y1 + y2)/2].isPicked == false){

					matrix[((x1 + x2)/2)*colum + (y1 + y2)/2].vort = interpolDotEdge(p1, p2);
					matrix[((x1 + x2)/2)*colum + (y1 + y2)/2].isPicked = true;
					matrix[((x1 + x2)/2)*colum + (y1 + y2)/2].layer = i-1;
					
				}
				else{

					float p5 = matrix[((x1 + x2)/2)*colum + (y1 + y2)/2].vort;
					float dot = interpolDotEdge(p1, p2);
					float dist = checkInterPol(p5, dot);
					matrix[((x1 + x2)/2)*colum + (y1 + y2)/2].vort = dot + dist;

				}

				if (matrix[((x1 + x4)/2)*colum + (y1 + y4)/2].isPicked == false){

					matrix[((x1 + x4)/2)*colum + (y1 + y4)/2].vort = interpolDotEdge(p1, p4);
					matrix[((x1 + x4)/2)*colum + (y1 + y4)/2].isPicked = true;
					matrix[((x1 + x4)/2)*colum + (y1 + y4)/2].layer = i-1;
					
				}
				else{

					float p5 = matrix[((x1 + x4)/2)*colum + (y1 + y4)/2].vort;
					float dot = interpolDotEdge(p1, p4);
					float dist = checkInterPol(p5, dot);
					matrix[((x1 + x4)/2)*colum + (y1 + y4)/2].vort = dot + dist;					

				}


				if (matrix[((x3 + x4)/2)*colum + (y3 + y4)/2].isPicked == false){

					matrix[((x3 + x4)/2)*colum + (y3 + y4)/2].vort = interpolDotEdge(p3, p4);
					matrix[((x3 + x4)/2)*colum + (y3 + y4)/2].isPicked = true;
					matrix[((x3 + x4)/2)*colum + (y3 + y4)/2].layer = i-1;
					
				}
				else{

					float p5 = matrix[((x3 + x4)/2)*colum + (y3 + y4)/2].vort;
					float dot = interpolDotEdge(p3, p4);
					float dist = checkInterPol(p5, dot);
					matrix[((x3 + x4)/2)*colum + (y3 + y4)/2].vort = dot + dist;
										
				}

				if (matrix[((x3 + x2)/2)*colum + (y3 + y2)/2].isPicked == false){

					matrix[((x3 + x2)/2)*colum + (y3 + y2)/2].vort = interpolDotEdge(p3, p2);
					matrix[((x3 + x2)/2)*colum + (y3 + y2)/2].isPicked = true;
					matrix[((x3 + x2)/2)*colum + (y3 + y2)/2].layer = i-1;
					
				}
				else{

					float p5 = matrix[((x3 + x2)/2)*colum + (y3 + y2)/2].vort;
					float dot = interpolDotEdge(p3, p2);
					float dist = checkInterPol(p5, dot);
					matrix[((x3 + x2)/2)*colum + (y3 + y2)/2].vort = dot + dist;
										
				}


				if (matrix[((x1 + x2 + x3 + x4)/4)*colum + (y1 + y2 + y3 + y4)/4].isPicked == false){

					matrix[((x1 + x2 + x3 + x4)/4)*colum + (y1 + y2 + y3 + y4)/4].vort = interpolDotMiddle(p1, p2, p3, p4);
					matrix[((x1 + x2 + x3 + x4)/4)*colum + (y1 + y2 + y3 + y4)/4].isPicked = true;
					matrix[((x1 + x2 + x3 + x4)/4)*colum + (y1 + y2 + y3 + y4)/4].layer = i-1;
					
					
				}
				else{

					float p5 = matrix[((x1 + x2 + x3 + x4)/4)*colum + (y1 + y2 + y3 + y4)/4].vort;
					float dot = interpolDotMiddle(p1, p2, p3, p4);
					float dist = checkInterPol(p5, dot);
					matrix[((x1 + x2 + x3 + x4)/4)*colum + (y1 + y2 + y3 + y4)/4].vort = dot + dist;
										
				}				
			}
		}	

		__syncthreads();
		
	}
}


//int main(){
void wavelet_decompression(Node* array, Node* matrix, int *countTrue){

	//Hämta in array med node-värden (skapa array)
	//int countTrue = 0;
	//Node *array;
	//int* countTrue;
	//countTrue = (int*) malloc(1*sizeof(int));
	//array = wavelet_start(countTrue, inMatrix);
	Node *d_array;
	const int size = row*colum* sizeof(Node);
	int len = *countTrue * sizeof (Node);
	//int layers = 4;

	std::cout<<sizeof(Node)<<std::endl;
	std::cout<<len<<std::endl;

	//const int row = array[0].x_index_global + 1;
	//const int colum = array[0].y_index_global + 1;
	//int size = row*colum* sizeof(Node);
				
	//Node* matrix;
	Node *d_matrix;	
	//matrix = (Node*) calloc(row*colum,sizeof(Node));

	/*int x = 0;	
	int y = 0;
	
	for(int i=0; i< (row*colum); i++){
		
			matrix[i].x_index_global = x;
			matrix[i].y_index_global = y;				
		
			
		if (y<colum - 1){

			y++;
		}
		else{

			y = 0;
			x++;
		}

			//std::cout<< /*"x = "<<matrix[i].x<<std::endl<< "y = "<<matrix[i].y<<std::endl<< "vort = "<<matrix[i].vort<<std::endl;/*<< "x_index = "<<matrix[i].x_index<<std::endl<< "y_index = "<<matrix[i].y_index<<std::endl<< "isPicked = "<<matrix[i].isPicked<<std::endl;*/
	//}
	

	if (hipMalloc(&d_matrix, size) != hipSuccess){

		std::cout<< "Can't allocate memory 1!"<<std::endl;
	}

	if (hipMalloc(&d_array, len) != hipSuccess){

		std::cout<< "Can't allocate memory 2!"<<std::endl;
	}

	if(hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice) != hipSuccess){

		std::cout<< "Could not copy to GPU 1!"<<std::endl;
		hipFree(d_matrix);
	}

	if(hipMemcpy(d_array, array, len, hipMemcpyHostToDevice) != hipSuccess){

		std::cout<< "Could not copy to GPU 2!"<<std::endl;
		hipFree(d_array);
	}


	//dim3 blockDim(*countTrue);
	dim3 blockDim(row, colum);
	dim3 gridDim(2, 2);


	wavelet_d_kernal<<<gridDim, blockDim>>>(d_matrix, d_array, row, colum, *countTrue, layers); //storlek på de som ska komma tillbaka, vaktor med sparade värden

	hipError_t err = hipDeviceSynchronize();
	std::cout<<"Run kernel: \n" << hipGetErrorString(err)<<std::endl;



	if(hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost) != hipSuccess){
		err = hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);
		std::cout<<"Copy to CPU: \n" << hipGetErrorString(err)<<std::endl;
		delete[] matrix;
		hipFree(d_matrix);
		std::cout<< "Can't copy back to CPU 1!"<<std::endl;

	}

	float vort;

	for(int y=colum-1; y>=0; y--){
    	
    	for(int x=0; x<row; x++){  
            
            vort = matrix[x*colum + y].vort;
            
            if(vort == 1){
                printf("1        ");
            }                       
                    
            else{
                printf ("%f ", vort);
            }
        }

     	std::cout<<std::endl;
    }

	float printIsPicked;
    
    for(int y=colum-1; y>=0; y--){
    	
    	for(int x=0; x<row; x++){            
                    
            printIsPicked = matrix[x*colum + y].isPicked;
            
            if(printIsPicked == 1){
                printf("1   ");
            }                       
                    
            else{
                printf ("0   ");
            }
        }

     	std::cout<<std::endl;
    }



	//hipFree(d_matrix);
	hipFree(d_array);


	delete[] array;

	//delete[] countTrue;

	//return 0;
	//std::cout<< d_matrix<<std::endl;
	//return d_matrix;
}