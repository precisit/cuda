#include "hip/hip_runtime.h"
#include <iostream>
#include "node.cpp"

 __device__ bool interpolDot(float p1, float p2, float p3, float p4, float p5, float tol){

 	float p;

 	p = (p1 + p2 + p3 + p4)/4.0;
 	p = p5 - p;
 	
 	if (p<0){

 		p = p*(-1);
 	}	

 	if (p > tol){

 		return true;
 	} 
 	else{return false;}
 }

__global__ void waveletkernal(Node* matrix, int row, int colum, float tol, int step, int layers){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;


	for (int i=0; i<layers; i++){

		if(idx < row -step && idy < colum -step){
		
			if (idx % 2 == 0 && idy % step == 0){

				float p1 = matrix[idx*colum + idy].vort;
				float p2 = matrix[idx*colum + idy + step].vort;
				float p3 = matrix[(idx + step)*colum + idy].vort;
				float p4 = matrix[(idx + step)*colum + idy + step].vort;
				float p5 = matrix[(idx + step/2)*colum + idy + step/2].vort;
			

				if (interpolDot(p1, p2, p3, p4, p5, tol) == true){

					matrix[(idx + step/2)*colum + idy + step/2].isPicked = true;
					matrix[(idx + step/2)*colum + idy + step/2].layer = i+1;
					
				}		
			}
					
		}
		step = step*2;
	}
}

int main(){

	const int row = 6;
	const int colum = 6;
	int step = 2;
	int layers = 2;	
	int size = row*colum* sizeof(Node);
	float tol = 0.1;
		
	Node* matrix;
	Node *d_matrix;	
	matrix = (Node*) calloc(row*colum,sizeof(Node));	

	int x = 0;	
	int y = 0;

	for(int i=0; i< (row*colum); i++){
		
			matrix[i].x_index = x;
			matrix[i].y_index = y;
			matrix[i].x = rand()%5;
			matrix[i].y = rand()%5;
			matrix[i].vort = rand()%10;
			//matrix[i].isPicked = true;

			if (y<colum - 1){

				y++;
			}
			else{

				y = 0;
				x++;
			}

			//std::cout<< /*"x = "<<matrix[i].x<<std::endl<< "y = "<<matrix[i].y<<std::endl<< */"vort = "<<matrix[i].vort<<std::endl;/*<< "x_index = "<<matrix[i].x_index<<std::endl<< "y_index = "<<matrix[i].y_index<<std::endl<< "isPicked = "<<matrix[i].isPicked<<std::endl;*/
		}
	

	if (hipMalloc(&d_matrix, size) != hipSuccess){

		std::cout<< "Can't allocate memory 1!"<<std::endl;
	}

	if(hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice) != hipSuccess){

		std::cout<< "Could not copy to GPU 1!"<<std::endl;
		hipFree(d_matrix);
	}

	dim3 blockDim(row, colum);
	dim3 gridDim(1, 1);

	waveletkernal<<<gridDim, blockDim>>>(d_matrix, row, colum, tol, step, layers); //storlek på de som ska komma tillbaka, vaktor med sparade värden

	hipError_t err = hipDeviceSynchronize();
	std::cout<<"Run kernel: \n" << hipGetErrorString(err)<<std::endl;

	if(hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost) != hipSuccess){
		//err = hipMemcpy(d_saveNode, saveNode, colum/step, hipMemcpyDeviceToHost);
		//std::cout<<"Copy to CPU: \n" << hipGetErrorString(err)<<std::endl;
		delete[] matrix;
		hipFree(d_matrix);
		std::cout<< "Can't copy back to CPU 3!"<<std::endl;

	}

	/*for(int i=0; i<row*colum; i++){

		std::cout<<"tja"<< /*matrix[i].x_index<<std::endl<< matrix[i].y_index<<std::endl<< matrix[i].x<<std::endl<< matrix[i].y<<std::endl<< matrix[i].vort<<std::endl<<matrix[i].isPicked<<std::endl;
	
	}*/

	float printVort;
    
    for(int x=0; x<row; x++){
                    
        for(int y=0; y<colum; y++){
            
            printVort = matrix[x + y*colum].vort;
            
            if(printVort == 0.0f){
                printf("0         ");
            }
                        
            else if(printVort>0.0f){
                printf ("%3f  ", printVort);
            }
                        
            else{
                printf ("%3f ", printVort);
            }
        }

     	std::cout<<std::endl;
    }

    float printIsPicked;
    
    for(int x=0; x<row; x++){
                    
        for(int y=0; y<colum; y++){
            
            printIsPicked = matrix[x + y*colum].isPicked;
            
            if(printIsPicked == 1){
                printf("true    ");
            }                       
                    
            else{
                printf ("false   ");
            }
        }

     	std::cout<<std::endl;
    }

   int countTrue = 0;

   for (int i=0; i<row; i++){

	   	for (int j=0; j<colum; j++){

	   		if (matrix[i*colum + j].isPicked == true){

	   			countTrue ++;
	   		}	    	
	    }
	}

	std::cout<<"countTrue: "<<countTrue<<std::endl;

	Node* ordedNodelist;
	ordedNodelist = (Node*) calloc(countTrue,sizeof(Node));
    
    int orderPlace = countTrue -1;

    //FIXA!!!!

	for (int m = 1; m <= layers; m++) {

		for (int i=0; i<row; i++){

	   		for (int j=0; j<colum; j++){
	   			
		    	if (matrix[i*colum + j].isPicked == true && matrix[i*colum + j].layer == m){

		    		ordedNodelist[orderPlace] = matrix[i*colum + j];

		    		orderPlace --;
		    	}
		    }
		}
	}

	for(int i=0; i<countTrue; i++){

		std::cout<<ordedNodelist[i].x_index<<std::endl<< ordedNodelist[i].y_index<<std::endl/*<< ordedNodelist[i].x<<std::endl<< ordedNodelist[i].y<<std::endl<< ordedNodelist[i].vort<<std::endl<<ordedNodelist[i].isPicked*/<<std::endl;
	}


	hipFree(d_matrix);
	delete[] matrix;	
	return 0;
}