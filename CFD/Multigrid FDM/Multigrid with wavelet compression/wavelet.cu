#include "hip/hip_runtime.h"
#include <iostream>
#include "node.cpp"

 __device__ bool interpolDot(float p1, float p2, float p3, float p4, float p5, float tol){

 	float p;

 	p = (p1 + p2 + p3 + p4)/4.0;
 	p = p5 - p;
 	
 	if (p<0){

 		p = p*(-1);
 	}	

 	if (p > tol){

 		return true;
 	} 
 	else{return false;}
 }

__global__ void waveletkernal(Node* matrix, int row, int colum, float tol, int step, int layers){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;


	for (int i=0; i<layers; i++){

		if(idx < row -step && idy < colum -step){
		
			if (idx % 2 == 0 && idy % step == 0){

				float p1 = matrix[idx*colum + idy].vort;
				float p2 = matrix[idx*colum + idy + step].vort;
				float p3 = matrix[(idx + step)*colum + idy].vort;
				float p4 = matrix[(idx + step)*colum + idy + step].vort;
				float p5 = matrix[(idx + step/2)*colum + idy + step/2].vort;
			

				if (interpolDot(p1, p2, p3, p4, p5, tol) == true){

					matrix[(idx + step/2)*colum + idy + step/2].isPicked = true;
					matrix[(idx + step/2)*colum + idy + step/2].layer = i+1;
					
				}		
			}
					
		}
		step = step*2;
	}
}

void waveletCompression(Node* matrix, Node* d_matrix, const int row, const int col, const datatype tol, int step, int layers){

	const int size = row*col*sizeof(Node);

	if(hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice) != hipSuccess){

		std::cout<< "Could not copy to GPU 1!"<<std::endl;
		hipFree(d_matrix);
	}

	dim3 blockDim(row, col);
	dim3 gridDim(1, 1);

	waveletkernal<<<gridDim, blockDim>>>(d_matrix, row, col, tol, step, layers); //storlek på de som ska komma tillbaka, vektor med sparade värden

	hipError_t err = hipDeviceSynchronize();
	//std::cout<<"Run kernel: \n" << hipGetErrorString(err)<<std::endl;

	if(hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost) != hipSuccess){
		//err = hipMemcpy(d_saveNode, saveNode, colum/step, hipMemcpyDeviceToHost);
		//std::cout<<"Copy to CPU: \n" << hipGetErrorString(err)<<std::endl;
		delete[] matrix;
		hipFree(d_matrix);
		std::cout<< "Can't copy back to CPU 3!"<<std::endl;

	}

	/*for(int i=0; i<row*colum; i++){
		std::cout<<"tja"<< /*matrix[i].x_index<<std::endl<< matrix[i].y_index<<std::endl<< matrix[i].x<<std::endl<< matrix[i].y<<std::endl<< matrix[i].vort<<std::endl<<matrix[i].isPicked<<std::endl;
	
	}*/

	/*

	float printVort;
    
    for(int x=0; x<row; x++){
                    
        for(int y=0; y<col; y++){
            
            printVort = matrix[x + y*col].vort;
            
            if(printVort == 0.0f){
                printf("0         ");
            }
                        
            else if(printVort>0.0f){
                printf ("%3f  ", printVort);
            }
                        
            else{
                printf ("%3f ", printVort);
            }
        }

     	std::cout<<std::endl;
    }

    float printIsPicked;
    
    for(int x=0; x<row; x++){
                    
        for(int y=0; y<col; y++){
            
            printIsPicked = matrix[x + y*col].isPicked;
            
            if(printIsPicked == 1){
                printf("true    ");
            }                       
                    
            else{
                printf ("false   ");
            }
        }

     	std::cout<<std::endl;
    }

   int countTrue = 0;

   for (int i=0; i<row; i++){

	   	for (int j=0; j<col; j++){

	   		if (matrix[i*col + j].isPicked == true){

	   			countTrue ++;
	   		}	    	
	    }
	}

	std::cout<<"countTrue: "<<countTrue<<std::endl;

	Node* ordedNodelist;
	ordedNodelist = (Node*) calloc(countTrue,sizeof(Node));
    
    int orderPlace = countTrue -1;

    //FIXA!!!!

	for (int m = 1; m <= layers; m++) {

		for (int i=0; i<row; i++){

	   		for (int j=0; j<col; j++){
	   			
		    	if (matrix[i*col + j].isPicked == true && matrix[i*col + j].layer == m){

		    		ordedNodelist[orderPlace] = matrix[i*col + j];

		    		orderPlace --;
		    	}
		    }
		}
	}


	*/

}

int main(){

	const int row = 6;
	const int colum = 6;
	int step = 2;
	int layers = 2;	
	//int size = row*colum* sizeof(Node);
	float tol = 0.1;
		
	Node* matrix;
	Node *d_matrix;	
	matrix = (Node*) calloc(row*colum,sizeof(Node));	

	int x = 0;	
	int y = 0;

	for(int i=0; i< (row*colum); i++){
		
		matrix[i].x_index = x;
		matrix[i].y_index = y;
		matrix[i].x = rand()%5;
		matrix[i].y = rand()%5;
		matrix[i].vort = rand()%10;
		//matrix[i].isPicked = true;

		if (y<colum - 1){
			y++;
		}
		else{
			y = 0;
			x++;
		}
		//std::cout<< /*"x = "<<matrix[i].x<<std::endl<< "y = "<<matrix[i].y<<std::endl<< */"vort = "<<matrix[i].vort<<std::endl;/*<< "x_index = "<<matrix[i].x_index<<std::endl<< "y_index = "<<matrix[i].y_index<<std::endl<< "isPicked = "<<matrix[i].isPicked<<std::endl;*/
	}
	

	if (hipMalloc(&d_matrix, row*colum*sizeof(Node)) != hipSuccess){

		std::cout<< "Can't allocate memory 1!"<<std::endl;
	}

	waveletCompression(matrix, d_matrix, row, colum, tol, step, layers);


	hipFree(d_matrix);
	delete[] matrix;	
	return 0;
}